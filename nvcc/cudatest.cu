#include <iostream>
#include <memory>
#include <string>

#include <hip/hip_runtime.h>

int *pArgc = NULL;
char **pArgv = NULL;

int main(int argc, char **argv) {
  pArgc = &argc;
  pArgv = argv;

  printf("%s Starting...\n\n", argv[0]);
  printf(
      " CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }
}
